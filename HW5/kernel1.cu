#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

__global__ void mandelKernel(int *output, float lowerX, float lowerY, float stepX, float stepY, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x; 
    int thisY = blockIdx.y * blockDim.y + threadIdx.y; 

    int index = thisY * resX + thisX; // (j * width + i);

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;
    
    int iter = 0;
    for (; iter < maxIterations; ++iter)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    
    output[index] = iter;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    const int img_size_mem = resX * resY * sizeof(int);
    int *hst_img = (int *) malloc(img_size_mem);
    
    // Allocate device memory
    int *dev_img;
    hipMalloc(&dev_img, img_size_mem);

    // Copy data from host memory to device memory
    hipMemcpy(dev_img, hst_img, img_size_mem, hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid(resX / BLOCK_SIZE, resY / BLOCK_SIZE);

    mandelKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_img, lowerX, lowerY, stepX, stepY, resX, maxIterations);

    hipMemcpy(hst_img, dev_img, img_size_mem, hipMemcpyDeviceToHost);
    memcpy(img, hst_img, img_size_mem);

    free(hst_img);
    hipFree(dev_img);
}
